
#include <hip/hip_runtime.h>
//////////////////////////////////////////////////////////////////////////
// This file is part of openPSTD.                                       //
//                                                                      //
// openPSTD is free software: you can redistribute it and/or modify     //
// it under the terms of the GNU General Public License as published by //
// the Free Software Foundation, either version 3 of the License, or    //
// (at your option) any later version.                                  //
//                                                                      //
// openPSTD is distributed in the hope that it will be useful,          //
// but WITHOUT ANY WARRANTY; without even the implied warranty of       //
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the        //
// GNU General Public License for more details.                         //
//                                                                      //
// You should have received a copy of the GNU General Public License    //
// along with openPSTD.  If not, see <http://www.gnu.org/licenses/>.    //
//                                                                      //
//////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////
//
// File: cuda_kernels32.cu
// Author: Louis van Harten
// Purpose:
//     Contains 32-bit version of the Cuda kernels used in the GPU
//     acceleration of the spatial derivatives needed in the PSTD method.
//
//////////////////////////////////////////////////////////////////////////

__global__ void derifact_multiplication(float *matr, float *mati, float *vecr, float *veci,
                                        int fftlen, int fftnum) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    int matindex = index_y * fftlen + index_x; // mat should be a contiguous array
    // if N1%16>0, we're starting too many threads.
    // There is probably a better way to do this, but just eating the surplus
    // should work.
    if (matindex < fftlen * fftnum) {
        float matreal = matr[matindex];
        float matimag = mati[matindex];
        float vecreal = vecr[index_x];
        float vecimag = veci[index_x];

        matr[matindex] = matreal * vecreal - matimag * vecimag;
        mati[matindex] = matreal * vecimag + matimag * vecreal;
    }
}

__global__ void
pressure_window_multiplication(float *mr, float *mi, float *A, float *p1, float *p2, float *p3,
                               int winlen, int Ns1, int Ns2, int Ns3, int fftlen, int fftnum,
                               float R21, float R00, float R31,
                               float R10) // passing a few by value seems to be more efficient
                                          // than building an array first in pycuda
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (index_y < fftnum) { // eat the surplus
        int matindex = index_y * fftlen + index_x;

        float G = 1;
        if (index_x < winlen) {
            G = A[index_x];
        } else if (index_x > winlen + Ns2 - 1 && index_x < winlen * 2 + Ns2) {
            G = A[index_x - Ns2];
        }
        mi[matindex] = 0;
        if (index_x < winlen) {
            mr[matindex] = G * (R21 * p1[Ns1 * index_y + index_x - winlen + Ns1] +
                                R00 * p2[Ns2 * index_y + winlen - 1 - index_x]);
        } else if (index_x < winlen + Ns2) {
            mr[matindex] = p2[Ns2 * index_y + index_x - winlen];
        } else if (index_x < winlen * 2 + Ns2) {
            mr[matindex] = G * (R31 * p3[Ns3 * index_y + index_x - winlen - Ns2] +
                                R10 * p2[Ns2 * index_y + 2 * Ns2 + winlen - 1 - index_x]);
        } else {
            mr[matindex] = 0; // zero padding
        }
    }
}

__global__ void
velocity_window_multiplication(float *mr, float *mi, float *A, float *p1, float *p2, float *p3,
                               int winlen, int Ns1, int Ns2, int Ns3, int fftlen, int fftnum,
                               float R21, float R00, float R31,
                               float R10) // passing a few by value seems to be more efficient
                                          // than building an array first in pycuda
{
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (index_y < fftnum) { // eat the surplus
        int matindex = index_y * fftlen + index_x;

        float G = 1;
        if (index_x < winlen) {
            G = A[index_x];
        } else if (index_x > winlen + Ns2 - 1 && index_x < winlen * 2 + Ns2) {
            G = A[index_x - Ns2];
        }
        mi[matindex] = 0;
        if (index_x < winlen) {
            mr[matindex] = G * (R21 * p1[Ns1 * index_y + index_x - winlen + Ns1 - 1] +
                                R00 * p2[Ns2 * index_y + winlen - index_x]);
        } else if (index_x < winlen + Ns2) {
            mr[matindex] = p2[Ns2 * index_y + index_x - winlen];
        } else if (index_x < winlen * 2 + Ns2) {
            mr[matindex] = G * (R31 * p3[Ns3 * index_y + index_x - winlen - Ns2 + 1] +
                                R10 * p2[Ns2 * index_y + 2 * Ns2 + winlen - 2 - index_x]);
        } else {
            mr[matindex] = 0; // zero padding
        }
    }
}
