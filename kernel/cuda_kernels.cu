
#include <hip/hip_runtime.h>
//////////////////////////////////////////////////////////////////////////
// This file is part of openPSTD.                                       //
//                                                                      //
// openPSTD is free software: you can redistribute it and/or modify     //
// it under the terms of the GNU General Public License as published by //
// the Free Software Foundation, either version 3 of the License, or    //
// (at your option) any later version.                                  //
//                                                                      //
// openPSTD is distributed in the hope that it will be useful,          //
// but WITHOUT ANY WARRANTY; without even the implied warranty of       //
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the        //
// GNU General Public License for more details.                         //
//                                                                      //
// You should have received a copy of the GNU General Public License    //
// along with openPSTD.  If not, see <http://www.gnu.org/licenses/>.    //
//                                                                      //
//////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////
//
// File: cuda_kernels.cu
// Author: Louis van Harten
// Purpose:
//     Contains 64-bit version of the Cuda kernels used in the GPU
//     acceleration of the spatial derivatives needed in the PSTD method.
//
//////////////////////////////////////////////////////////////////////////

__global__ void derifact_multiplication(double *matr, double *mati, double *vecr, double *veci,
                                        int fftlen, int fftnum) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    int matindex = index_y * fftlen + index_x; // mat should be a contiguous array
    // if N1%16>0, we're starting too many threads.
    // There is probably a better way to do this, but just eating the surplus
    // should work.
    if (matindex < fftlen * fftnum) {
        double matreal = matr[matindex];
        double matimag = mati[matindex];
        double vecreal = vecr[index_x];
        double vecimag = veci[index_x];

        matr[matindex] = matreal * vecreal - matimag * vecimag;
        mati[matindex] = matreal * vecimag + matimag * vecreal;
    }
}

__global__ void pressure_window_multiplication(double *mr, double *mi, double *A, double *p1,
                                               double *p2, double *p3, int winlen, int Ns1, int Ns2,
                                               int Ns3, int fftlen, int fftnum, double R21,
                                               double R00, double R31, double R10) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (index_y < fftnum) { // eat the surplus
        int matindex = index_y * fftlen + index_x;

        double G = 1;
        if (index_x < winlen) {
            G = A[index_x];
        } else if (index_x > winlen + Ns2 - 1 && index_x < winlen * 2 + Ns2) {
            G = A[index_x - Ns2];
        }
        mi[matindex] = 0;
        if (index_x < winlen) {
            mr[matindex] = G * (R21 * p1[Ns1 * index_y + index_x - winlen + Ns1] +
                                R00 * p2[Ns2 * index_y + winlen - 1 - index_x]);
        } else if (index_x < winlen + Ns2) {
            mr[matindex] = p2[Ns2 * index_y + index_x - winlen];
        } else if (index_x < winlen * 2 + Ns2) {
            mr[matindex] = G * (R31 * p3[Ns3 * index_y + index_x - winlen - Ns2] +
                                R10 * p2[Ns2 * index_y + 2 * Ns2 + winlen - 1 - index_x]);
        } else {
            mr[matindex] = 0; // zero padding
        }
    }
}

__global__ void velocity_window_multiplication(double *mr, double *mi, double *A, double *p1,
                                               double *p2, double *p3, int winlen, int Ns1, int Ns2,
                                               int Ns3, int fftlen, int fftnum, double R21,
                                               double R00, double R31, double R10) {
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (index_y < fftnum) { // eat the surplus
        int matindex = index_y * fftlen + index_x;

        double G = 1;
        if (index_x < winlen) {
            G = A[index_x];
        } else if (index_x > winlen + Ns2 - 1 && index_x < winlen * 2 + Ns2) {
            G = A[index_x - Ns2];
        }
        mi[matindex] = 0;
        if (index_x < winlen) {
            mr[matindex] = G * (R21 * p1[Ns1 * index_y + index_x - winlen + Ns1 - 1] +
                                R00 * p2[Ns2 * index_y + winlen - index_x]);
        } else if (index_x < winlen + Ns2) {
            mr[matindex] = p2[Ns2 * index_y + index_x - winlen];
        } else if (index_x < winlen * 2 + Ns2) {
            mr[matindex] = G * (R31 * p3[Ns3 * index_y + index_x - winlen - Ns2 + 1] +
                                R10 * p2[Ns2 * index_y + 2 * Ns2 + winlen - 2 - index_x]);
        } else {
            mr[matindex] = 0; // zero padding
        }
    }
}
